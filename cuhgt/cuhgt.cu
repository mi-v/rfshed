#include "hip/hip_runtime.h"
#include <stdint.h>
#include <unistd.h>
#include <algorithm>
#include "cuhgt.h"

#define cuErr(call)  {hipError_t err; if (hipSuccess != (err=(call))) throw cuErrX{err, hipGetErrorString(err), __FILE__, __LINE__};}

__global__ void byteswap16(int32_t* Hgt) {
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;

    if (row < 1201 && col < 602) {
        int ofs = row * 640 + col;
        Hgt[ofs] = ((Hgt[ofs] >> 8) & 0x00ff00ff) | ((Hgt[ofs] << 8) & 0xff00ff00);
    }
}

void* Hgt_ha;
void* Hgt_hb;
hipEvent_t *xfer_a = new hipEvent_t;
hipEvent_t *xfer_b = new hipEvent_t;

extern "C" {
    UploadResult uploadHgt(int fd) {
        UploadResult Res = {0};
        short* Hgt_d;
        try {
            cuErr(hipMalloc(&Hgt_d, 1280 * 1201 * sizeof(short)));
            hipEventSynchronize(*xfer_a);
            read(fd, Hgt_ha, HGTSIZE);
            cuErr(hipMemcpy2DAsync(
                Hgt_d,
                1280 * sizeof(short),
                Hgt_ha,
                1201 * sizeof(short),
                1201 * sizeof(short),
                1201,
                hipMemcpyHostToDevice
            ));
            cuErr(hipEventRecord(*xfer_a));
            byteswap16<<<dim3(19, 38), dim3(32, 32), 0>>>(reinterpret_cast<int32_t*>(Hgt_d));
            std::swap(xfer_a, xfer_b);
            std::swap(Hgt_ha, Hgt_hb);
            Res.ptr = (uint64_t)Hgt_d;
        } catch (cuErrX error) {
            Res.error = error;
        }
        return Res;
    }

    void freeHgt(uint64_t ptr) {
        hipFree((short*)ptr);
    }

    cuErrX cuhgtInit() {
        cuErrX err = {0};
        try {
            cuErr(hipHostAlloc(&Hgt_ha, HGTSIZE, hipHostMallocWriteCombined));
            cuErr(hipHostAlloc(&Hgt_hb, HGTSIZE, hipHostMallocWriteCombined));
            cuErr(hipEventCreateWithFlags(xfer_a, hipEventDisableTiming));
            cuErr(hipEventCreateWithFlags(xfer_b, hipEventDisableTiming));
        } catch (cuErrX error) {
            err = error;
        }
        return err;
    }
}
