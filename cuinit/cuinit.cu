#include "cuinit.h"

extern "C" {
    void cuinitInit() {
        hipSetDevice(0);
        hipSetDeviceFlags(hipDeviceScheduleBlockingSync | hipDeviceMapHost);
    }
}
